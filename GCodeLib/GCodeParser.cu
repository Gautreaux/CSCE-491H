
#include <fstream>
#include <sstream>
#include <vector>

#include "GCodeParser.h"
#include "../pch.h"

GCodeParser::GCodeParser(const std::string filePath) : errorFlags(0), fileSize(0), filePath(filePath) {
    // setup the object

    // do the file parsing
    bool valid = parseFile(filePath);
#ifdef DEBUG
    printf("Parsing %s for file %s\n", valid ? "successful" : "failed", filePath.c_str());
#endif
    if(valid == false){
        setErrorBit(ErrorTypes::PARSE_ERROR);
        return;
    }

    // simple checks
    //  this could be more efficient, but whatever
    //  the checks do a decent amount of duplicate work
    if(isMonotonicIncreasingZ() == false){
        setErrorBit(ErrorTypes::MONO_ERROR);
        valid = false;
    }
    if(isZSlicedPrint() == false){
        setErrorBit(ErrorTypes::Z_ERROR);
        valid = false;
    }
    if(isContinuousPrint() == false){
        setErrorBit(ErrorTypes::CONTINUOUS_ERROR);
        valid = false;
    }
#ifdef DEBUG
    printf("Post-Parse Checks: %s\n", valid ? "Passing" : "Failed");
#endif      
    if(valid == false){
#ifdef DEBUG
        // not the most efficient, but shouldn't be run so thats cool
        //  also, the compiler shouldn't recognize/cache results?
        // TODO - rewrite to use the error bits and problem is solved
        printf("\tMonotonic increasing z: %s\n", isMonotonicIncreasingZ() ? "Passing" : "Failed");
        printf("\tZ-Sliced Print: %s\n", isZSlicedPrint() ? "Passing" : "Failed");
        printf("\tContinuous Print: %s\n", isContinuousPrint() ? "Passing" : "Failed");
#endif
        return;
    }


    // build the zLayers
    assert(readErrorBit(ErrorTypes::Z_ERROR) == false);
    zLayers.empty();
    zLayers.push_back(segmentsList[0].getStartPoint().getZ());
    for(auto segment : segmentsList){
        if(segment.isPrintSegment()){
            assert(segment.isZParallel()); // implied by Z_ERROR bit
            if(zLayers.size() == 0 || (segment.getStartPoint().getZ() > zLayers.back())){ 
                zLayers.push_back(segment.getStartPoint().getZ());
            }
        }
    }

    //now we want to split the print segments 
    originalSegmentsList = std::move(segmentsList);
    for(auto& originalSeg : originalSegmentsList){
        if(originalSeg.isPrintSegment() == false){
            segmentsList.push_back(originalSeg);
            continue;
        }
        if(originalSeg.length() <= SPLIT_TARGET_MM){
            //this segment is already short enough, so skip it
            segmentsList.push_back(originalSeg);
            continue;
        }
        
        double origLen = originalSeg.length();
        //check, should be guaranteed above
        assert(originalSeg.isZParallel());

        const Vector3 stepAmount = originalSeg.getSlope().scaleVector(SPLIT_TARGET_MM);
        const Point3& startPoint = originalSeg.getStartPoint();
        Point3 lastPoint = startPoint;
        double printPerUnit = originalSeg.getPrintAmount() / origLen;
        
        do{
            Point3 newPoint = lastPoint + stepAmount;
            if(getPointDistance(startPoint, newPoint) > origLen){
                //last segment
                double thisLen = getPointDistance(lastPoint, originalSeg.getEndPoint());
                segmentsList.push_back(GCodeSegment(lastPoint, originalSeg.getEndPoint(), thisLen*printPerUnit));
                break;
            }else{
                //partial segment
                assert(DOUBLE_EQUAL(getPointDistance(lastPoint, newPoint), SPLIT_TARGET_MM));
                segmentsList.push_back(GCodeSegment(lastPoint, newPoint, printPerUnit*SPLIT_TARGET_MM));
                lastPoint = newPoint;
            }
        }while(true);
    }
}

bool GCodeParser::parseFile(const std::string filePath){
    std::ifstream infile(filePath);
    if(infile.is_open() == false){
#ifdef DEBUG
        printf("File for path %s failed to open.\n", filePath.c_str());
#endif
        return false;
#ifdef DEBUG
    }else{
        printf("File for path %s opened successfully\n", filePath.c_str());
#endif
    }

    //seems like this shouldn't be necessary
    infile.clear();
    infile.seekg(0, std::ios::beg);
    auto b = infile.tellg();
    infile.seekg(0, std::ios::end);
    fileSize = infile.tellg() - b;

    if(fileSize > (MAX_FILE_SIZE_MB * MB_TO_B)){
        setErrorBit(ErrorTypes::FILE_TOO_LARGE);
        return false;
    }

    //return to beginning
    infile.seekg(0, std::ios::beg);

    //do the meat of the processing
    std::string line;
    std::vector<std::string> lineTokens;
    lineTokens.reserve(20); // attempt to prevent resizing later
#ifdef DEBUG
    int lineCounter = 0; //so we can see the problematic line in gdb
#endif

    GCodeUnits currentUnits = GCodeUnits::MILLIMETERS; //TODO - should probably start as some "not-set" until file provides a unit
    GCodePositioningType currentPositioningType = GCodePositioningType::ABSOLUTE; // TODO ^
    double posX = 0; // TODO ^
    double posY = 0; // TODO ^
    double posZ = 0; // TODO ^
    double posE = 0; // TODO ^

    while (std::getline(infile, line)){
#ifdef DEBUG
        lineCounter++;
#endif
        if(line == ""){
            continue;
        }

        std::istringstream lineStream(line);
        std::string token;
        lineTokens.clear(); // remove the previous line's tokens

        //break the line apart into its tokens
        while(lineStream >> token){
            if(token == ";"){
                //a comment has been started,
                //  the rest of the line will be ignored, so we can skip it
                break;
            }
            
            lineTokens.push_back(token);

        }

        if(lineTokens.size() == 0){
            //this entire line is a comment
            continue;
        }

        //now we will process on the various tokens
        auto split = splitToken(lineTokens[0]);
        if (split.letter == 'G'){
            // some form of a move command
            if (split.number == 20){
                // G20 - switch units type to inches
                currentUnits = GCodeUnits::INCHES;
                throw UnrecognizedCommandException(line, lineTokens[0]);
            }else if (split.number == 21){
                // G21 - switch units type to millimeters
                currentUnits = GCodeUnits::MILLIMETERS;
            }else if (split.number == 90){
                // G90 - switch positioning type to absolute
                currentPositioningType = GCodePositioningType::ABSOLUTE;
            }else if (split.number == 91){
                // G91 - switch positioning type to relative
                currentPositioningType = GCodePositioningType::RELATIVE;
            }else if (split.number == 28){
                // G28 - home axis
                //  with no arguments, home all the axis
                //  with arguments, home the ones specified
                if(lineTokens.size() == 1){
                    // no arguments, home all
                    posX = 0;
                    posY = 0;
                    posZ = 0;
                    posE = 0;
                }
                else{
                    for(unsigned int i = 1; i < lineTokens.size(); i++){
                        //TODO - i think that you can home axis to positions
                        //  line "G28 X5" is valid
                        auto st = splitToken(lineTokens[i]);
                        if(st.letter == 'X'){
                            posX = 0;
                        }else if(st.letter == 'Y'){
                            posY = 0;
                        }else if(st.letter == 'Z'){
                            posZ = 0;
                        }else if(st.letter == 'E'){
                            posE = 0;
                        }
                        else{
                            //just ignore any other axis homing that may need to occur
                            // such as the F axis
                        }
                    }
                }
            }else if(split.number == 1){
                if(currentUnits == GCodeUnits::INCHES){
                    //this should be more descriptive but whatever
                    throw UnrecognizedCommandException(line, lineTokens[0]);
                }

                //G1 - standard liner interpolation move
                double startX = posX;
                double startY = posY;
                double startZ = posZ;
                double startE = posE;

                for(unsigned int i = 1; i < lineTokens.size(); i++){
                    auto st = splitToken(lineTokens[i]);
                    if(st.letter == 'X'){
                        posX = (currentPositioningType == GCodePositioningType::ABSOLUTE) ? st.number : posX + st.number;
                    }else if(st.letter == 'Y'){
                        posY = (currentPositioningType == GCodePositioningType::ABSOLUTE) ? st.number : posY + st.number;
                    }if(st.letter == 'Z'){
                        posZ = (currentPositioningType == GCodePositioningType::ABSOLUTE) ? st.number : posZ + st.number;
                    }if(st.letter == 'E'){
                        posE = (currentPositioningType == GCodePositioningType::ABSOLUTE) ? st.number : posE + st.number;
                    }else{
                        // nothing to do on this input type
                        //  EX: F - setting the feed rate to 
                    }
                }

                //construct the actual line segment object
                Point3 startPoint(startX, startY, startZ);
                Point3 endPoint(posX, posY, posZ);
                double printAmount = posE - startE;

                if(startPoint == endPoint){
                    //no-move command
                    //just pass without adding this as a segment
                }
                else{
                    //prune the segments list to the first print segment
                    GCodeSegment thisSeg(startPoint, endPoint, printAmount);
                    if(thisSeg.isPrintSegment() || segmentsList.size() > 0){
                        segmentsList.push_back(thisSeg);
                    }
                    //segmentsList.emplace_back(startPoint, endPoint, printAmount);
                }
                
            }else{
                //this was an unrecognized command, so we give up
                throw UnrecognizedCommandException(line, lineTokens[0]);
            }
        }else if (split.letter == 'M'){
            // a M command,
            // for the moment we just skip these
            // probably dont want to just blanket ignore, but it seems to be fine
        }else{
            //this was an unrecognized command, so we give up
            throw UnrecognizedCommandException(line, lineTokens[0]);
        }
        
    }

    //ifstream destructor should call this automatically?
    infile.close();

    //parsing completes successfully
    return true;
}

GCodeParser::SplitToken GCodeParser::splitToken(const std::string token){
    GCodeParser::SplitToken toReturn;
    toReturn.letter = toupper(token[0]);
    toReturn.number = std::stod(token.substr(1));
    return toReturn;
}

GCodeParser::UnrecognizedCommandException::UnrecognizedCommandException(const std::string line, const std::string token):
        problemLine(line), problemToken(token) 
{
};

bool GCodeParser::isMonotonicIncreasingZ(void) const {
    double lastZ = segmentsList[0].getStartPoint().getZ();

    for(auto segment : segmentsList){
        if(lastZ > segment.getStartPoint().getZ()){
            return false;
        }else if(lastZ < segment.getStartPoint().getZ()){
            lastZ = segment.getStartPoint().getZ();
        }

        if(lastZ > segment.getEndPoint().getZ()){
            return false;
        }else if(lastZ < segment.getEndPoint().getZ()){
            lastZ = segment.getEndPoint().getZ();
        }
    }
    return true;
}

bool GCodeParser::isZSlicedPrint(void) const {
    for(auto segment : segmentsList){
        if(segment.isPrintSegment() == true && segment.isZParallel() == false){
            return false;
        }
    }
    return true;
}

bool GCodeParser::isContinuousPrint(void) const {
    Point3 lastPos = segmentsList[0].getStartPoint();

    for(auto segment : segmentsList){
        if(segment.getStartPoint() != lastPos){
            return false;
        }
        lastPos = segment.getEndPoint();
    }
    return true;
}

unsigned char GCodeParser::convertParseValid(GCodeParser::ErrorTypes p) const {
    switch (p)
    {
    case ErrorTypes::PARSE_ERROR:
        return 0;
    case ErrorTypes::MONO_ERROR:
        return 1;
    case ErrorTypes::Z_ERROR:
        return 2;
    case ErrorTypes::CONTINUOUS_ERROR:
        return 3;
    case ErrorTypes::FILE_TOO_LARGE:
        return 4;
    }
    throw std::exception();
};

const GCodeSegment& GCodeParser::at(unsigned int i) const{
    if(i < 0 || i > numberSegments()){
        throw std::out_of_range("");
    }

    return segmentsList[i];
}

const GCodeSegment& GCodeParser::orig_at(unsigned int i) const{
    if(i < 0 || i > numberOrigSegments()){
        throw std::out_of_range("");
    }

    return segmentsList[i];
}

unsigned int GCodeParser::getLayerStartIndex(double zLayerTarget, unsigned int hint) const {
    //TODO - could be a binary search or something more efficient, because
    //requires segments to be in monotonic increasing order
    assert(readErrorBit(ErrorTypes::MONO_ERROR) == false);

    for(unsigned int i = hint; i < segmentsList.size(); i++){
        const GCodeSegment& gcs = segmentsList.at(i);
        if(gcs.isZParallel()){
            double z = gcs.getStartPoint().getZ();
            if(DOUBLE_EQUAL(z, zLayerTarget)){
                return i;
            }else if(z > zLayerTarget){
                break;
            }
        }
    }

    throw std::runtime_error("Could not match provided zLayer to one in file");
}


unsigned int GCodeParser::getLayerEndIndex(double zLayerTarget, unsigned int hint) const {
    //TODO - could be a binary search or something more efficient, because
    //requires segments to be in monotonic increasing order
    assert(readErrorBit(ErrorTypes::MONO_ERROR) == false);

    hint = ((hint < segmentsList.size()-1) ? hint : (segmentsList.size() - 1));

    for(int i = hint; i >= 0; i--){
        const GCodeSegment& gcs = segmentsList.at(i);
        if(gcs.isZParallel()){
            auto z = gcs.getStartPoint().getZ();
            if(z == zLayerTarget){
                return i;
            }else if(z < zLayerTarget){
                break;
            }
        }
    }

    throw std::runtime_error("Could not match provided zLayer to one in file");
}

unsigned int GCodeParser::getLayerOrigStartIndex(double zLayerTarget, unsigned int hint) const {
    //TODO - could be a binary search or something more efficient, because
    //requires segments to be in monotonic increasing order
    assert(readErrorBit(ErrorTypes::MONO_ERROR) == false);

    for(unsigned int i = hint; i < originalSegmentsList.size(); i++){
        const GCodeSegment& gcs = originalSegmentsList.at(i);
        if(gcs.isZParallel()){
            auto z = gcs.getStartPoint().getZ();
            if(z == zLayerTarget){
                return i;
            }else if(z > zLayerTarget){
                break;
            }
        }
    }

    throw std::runtime_error("Could not match provided zLayer to one in file");
}


unsigned int GCodeParser::getLayerOrigEndIndex(double zLayerTarget, unsigned int hint) const {
    //TODO - could be a binary search or something more efficient, because
    //requires segments to be in monotonic increasing order
    assert(readErrorBit(ErrorTypes::MONO_ERROR) == false);

    hint = ((hint < originalSegmentsList.size()-1) ? hint : (originalSegmentsList.size() - 1));

    for(int i = hint; i >= 0; i--){
        const GCodeSegment& gcs = originalSegmentsList.at(i);
        if(gcs.isZParallel()){
            auto z = gcs.getStartPoint().getZ();
            if(z == zLayerTarget){
                return i;
            }else if(z < zLayerTarget){
                break;
            }
        }
    }

    throw std::runtime_error("Could not match provided zLayer to one in file");
}