#include "ChainLayerMetaAccelerator.cuh"

NVCC_D bool checkCollisions(const LineSegment* const segmentsList,
    unsigned int a1Index, unsigned int a2Index)
{
    // if(a1Index == 0 && a2Index == 4){
    //     bool b = DOUBLE_GEQ(segmentsList[a1Index].minSeperationDistance(segmentsList[a2Index]), 25.0);
    //     printf("CUDA i = 0, j = 4: %d\n", b);
    // }
    return DOUBLE_GEQ(segmentsList[a1Index].minSeperationDistance(segmentsList[a2Index]), 25.0);
}

NVCC_G void precacheChains(char* const bitTable, const LineSegment* const segmentsList,
    const unsigned int segmentsQty)
{
    const unsigned int mySegentIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if(mySegentIndex >= segmentsQty){
        return;
    }
    // printf("Starting index: %d\n", mySegentIndex);
    unsigned int ctr = 0;
    for(unsigned int thisRow = 0; thisRow < CEIL_DIVISION(segmentsQty, sizeof(char)*8); thisRow++){
        char c = 0;
        for(unsigned int j = 0; j < sizeof(char)*8; j++, ctr++){
            if(ctr >= segmentsQty){
                continue;
            }

            // printf("CALC: %d %d\n", mySegentIndex, ctr);

            bool b = checkCollisions(segmentsList, mySegentIndex, ctr);
            c |= (((b) ? 1 : 0) << j);

            // if(mySegentIndex == 0 && (ctr == 4 || ctr == 32 || ctr == 162 || ctr == 196)){
            //     printf("CUDA err: i = %d, j = %d, offset = %d, bool: %d, segmentsQty: %d, thisRow: %d,"
            //             "mySegmentIndex: %d, storeIndex: %d, ctr: %d \n",
            //             mySegentIndex, ctr, j, b, segmentsQty, thisRow, 
            //             mySegentIndex, segmentsQty*thisRow + mySegentIndex,
            //             ctr);
            // }
        }
        // int d = (int)c;
        // printf("CUDA set: %d %d\n", segmentsQty*thisRow + mySegentIndex, d);
        bitTable[segmentsQty*thisRow + mySegentIndex] = c;
    }
}

PreCache offloadPrecaching(
    const unsigned int numberPrintSegments,
    const std::vector<LineSegment>& segmentsList
){
    assert(numberPrintSegments == segmentsList.size());

    const unsigned int numberRows = CEIL_DIVISION(numberPrintSegments, sizeof(char)*8);

    LineSegment* segmentsList_device;
    const unsigned int sizeofSegListBytes = sizeof(LineSegment)*numberPrintSegments;

    char* outputList_device;
    const unsigned int sizeofOutputBytes = numberRows * numberPrintSegments;

    std::cout << "Attempting hipMalloc of segments/output: " << sizeofSegListBytes << " " << sizeofOutputBytes << std::endl;


    for(unsigned int i = 0; i < 2 ; i++){
        auto e = hipMalloc(
            ((i == 0) ? ((void**)&segmentsList_device) : (void**)&outputList_device),
            ((i == 0) ? sizeofSegListBytes : sizeofOutputBytes)
        );
        if(e == hipErrorOutOfMemory){
            std::cout << "Device OOM?" << std::endl;
        }
        if(e != hipSuccess){
            std::cout << "Error occurred in CUDA malloc: " << e << std::endl;
            printf("Reason: %s\n", hipGetErrorString(e));
            exit(e);
        };
    }

    hipMemcpy(segmentsList_device, segmentsList.data(), sizeofSegListBytes, hipMemcpyHostToDevice);
    hipMemset(outputList_device, 0, sizeofOutputBytes);

    const unsigned int threadsPerBlock = 256;
    const unsigned int numberBlocks =  CEIL_DIVISION(numberPrintSegments, threadsPerBlock);

    precacheChains<<<numberBlocks, threadsPerBlock>>>(outputList_device, segmentsList_device, numberPrintSegments);

    auto e = hipDeviceSynchronize();
    if(e != hipSuccess){
        printf("CUDA synchronize failed with %d:%s\n",
            e, hipGetErrorString(e));
        exit(e);
    }

    char* const outputList = (char*)malloc(sizeofOutputBytes);
    hipMemcpy(outputList, outputList_device, sizeofOutputBytes, hipMemcpyDeviceToHost);

    // for(unsigned int i = 0; i < sizeofOutputBytes; i++){
    //     int k;
    //     k = (int)(outputList[i]);
    //     printf("HOST read: %d %d\n", i, k);
    // }

    hipFree(segmentsList_device);
    hipFree(outputList_device);

    return PreCache(outputList, numberPrintSegments);
}

void logCUDAInfo(void){
    std::cout << "=============================" << std::endl;
    std::cout << "CUDA info:" << std::endl;
    
    int driverVersion, runtimeVersion, deviceCount;

    auto e = hipRuntimeGetVersion(&runtimeVersion);
    if(e != 0){
        printf("get runtime CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    std::cout << "CUDA Runtime Version: " << runtimeVersion << std::endl;
    e = hipDriverGetVersion(&driverVersion);
    if(e != 0){
        printf("get version CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    std::cout << "CUDA Driver Version: " << driverVersion << std::endl;
    e = hipGetDeviceCount(&deviceCount);
    if(e != 0){
        printf("get device CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    std::cout << "CUDA Device Count: " << deviceCount << std::endl;

    printf("!!!REMOVE ME!!!\n");
    e = hipDeviceSetLimit(hipLimitPrintfFifoSize, 1*1024*1024*1024);
    if(e != 0){
        printf("set limit CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    size_t mySize;
    hipDeviceGetLimit(&mySize, hipLimitPrintfFifoSize);
    std::cout << "Resolved printf size: " << mySize << std::endl;

    std::cout << "=============================" << std::endl;
}

PreCache::PreCache(void) : c(nullptr), size(0)
{};

PreCache::PreCache(const char* const c, const unsigned int s) :
c(c), size(s)
{};

PreCache& PreCache::operator=(PreCache&& other){
    if(c != nullptr){
        free((void*)c);
        c = nullptr;
        size = 0;
    }
    size = other.size;
    c = other.c;

    other.c = nullptr;
    other.size = 0;

    return *this;
}

PreCache::~PreCache(void){
    if(c!=nullptr){
        free((void*)c);
        c = nullptr;
        size = 0;
    }
}