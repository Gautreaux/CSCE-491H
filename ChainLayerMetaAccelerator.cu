#include "ChainLayerMetaAccelerator.cuh"

NVCC_D bool checkCollisions(const LineSegment* const segmentsList,
    unsigned int a1Index, unsigned int a2Index)
{
    return DOUBLE_GEQ(segmentsList[a1Index].minSeperationDistance(segmentsList[a2Index]), 25.0);
}

NVCC_G void precacheChains(char* const bitTable, const LineSegment* const segmentsList,
    const unsigned int segmentsQty, const unsigned int rowWidth)
{
    const unsigned int mySegentIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int ctr = 0;
    for(unsigned int thisRow = 0; thisRow < CEIL_DIVISION(segmentsQty, sizeof(char)); thisRow++){
        char c = 0;
        for(unsigned int j = 0; j < sizeof(char); j++, ctr++){
            if(ctr >= segmentsQty){
                continue;
            }
            bool b = checkCollisions(segmentsList, mySegentIndex, ctr);
            c |= (((b) ? 1 : 0) << j);
        }
        bitTable[rowWidth*thisRow + mySegentIndex] = c;
    }
}

template<typename CLM_Type>
void offloadPrecaching(CLM_Type* const clm){
    const unsigned int numberPrintSegments = clm->getNumberPrintSegmentsInLayer();
    const unsigned int rowWidth = CEIL_DIVISION(numberPrintSegments, sizeof(char));
    std::vector<LineSegment> segmentsList;
    segmentsList.reserve(numberPrintSegments);
    for(unsigned int i = 0; i < numberPrintSegments; i++){
        segmentsList.push_back(clm->getSegmentByLayerIndex(i));
    }

    LineSegment* segmentsList_device;
    const unsigned int sizeofSegListBytes = sizeof(LineSegment)*numberPrintSegments;

    auto e = hipMalloc((void**)&segmentsList_device, sizeofSegListBytes);
    if(e == hipErrorOutOfMemory){
        std::cout << "Device OOM" << std::endl;
    }
    if(e != hipSuccess){
        std::cout << "Error occurred in CUDA malloc: " << e << std::endl;
        exit(e);
    };

    hipMemcpy(segmentsList_device, segmentsList.data(), sizeofSegListBytes, hipMemcpyHostToDevice);




    hipFree(segmentsList_device);
}
