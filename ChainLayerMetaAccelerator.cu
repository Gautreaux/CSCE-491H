#include "hip/hip_runtime.h"
#include "pch.h"
#include "GeometryLib/LineSegment.cu"


__device__ __inline__ bool checkCollisions(const LineSegment* const segmentsList,
    unsigned int a1Index, unsigned int a2Index)
{
    return DOUBLE_GEQ(segmentsList[a1Index].minSeperationDistance(segmentsList[a2Index]), 25.0);
}

__global__ void precacheChains(char* const bitTable, const LineSegment* const segmentsList,
    const unsigned int segmentsQty, const unsigned int rowWidth)
{
    const unsigned int mySegentIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int ctr = 0;
    for(unsigned int thisRow = 0; thisRow < CEIL_DIVISION(segmentsQty, sizeof(char)); thisRow++){
        char c = 0;
        for(unsigned int j = 0; j < sizeof(char); j++, ctr++){
            if(ctr >= segmentsQty){
                continue;
            }
            bool b = checkCollisions(segmentsList, mySegentIndex, ctr);
            c |= (((b) ? 1 : 0) << j);
        }
        bitTable[rowWidth*thisRow + mySegentIndex] = c;
    }
}