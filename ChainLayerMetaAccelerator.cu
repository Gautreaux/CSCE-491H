#include "ChainLayerMetaAccelerator.cuh"

NVCC_D bool checkCollisions(const LineSegment* const segmentsList,
    unsigned int a1Index, unsigned int a2Index)
{
    return DOUBLE_GEQ(segmentsList[a1Index].minSeperationDistance(segmentsList[a2Index]), 25.0);
}

NVCC_G void precacheChains(char* const bitTable, const LineSegment* const segmentsList,
    const unsigned int segmentsQty, const unsigned int rowWidth)
{
    const unsigned int mySegentIndex = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int ctr = 0;
    for(unsigned int thisRow = 0; thisRow < CEIL_DIVISION(segmentsQty, sizeof(char)); thisRow++){
        char c = 0;
        for(unsigned int j = 0; j < sizeof(char); j++, ctr++){
            if(ctr >= segmentsQty){
                continue;
            }
            bool b = checkCollisions(segmentsList, mySegentIndex, ctr);
            c |= (((b) ? 1 : 0) << j);
        }
        bitTable[rowWidth*thisRow + mySegentIndex] = c;
    }
}

void offloadPrecaching(
    const unsigned int numberPrintSegments,
    const std::vector<LineSegment>& segmentsList
){
    assert(numberPrintSegments == segmentsList.size());

    const unsigned int rowWidth = CEIL_DIVISION(numberPrintSegments, sizeof(char));

    LineSegment* segmentsList_device;
    const unsigned int sizeofSegListBytes = sizeof(LineSegment)*numberPrintSegments;

    char* outputList_device;
    const unsigned int sizeofOutputBytes = rowWidth * numberPrintSegments;

    std::cout << "Attempting hipMalloc of segments/output: " << sizeofSegListBytes << " " << sizeofOutputBytes << std::endl;


    for(unsigned int i = 0; i < 2 ; i++){
        auto e = hipMalloc(
            ((i == 0) ? ((void**)&segmentsList_device) : (void**)&outputList_device),
            ((i == 0) ? sizeofSegListBytes : sizeofOutputBytes)
        );
        if(e == hipErrorOutOfMemory){
            std::cout << "Device OOM" << std::endl;
        }
        if(e != hipSuccess){
            std::cout << "Error occurred in CUDA malloc: " << e << std::endl;
            printf("%s\n", hipGetErrorString(e));
            exit(e);
        };
    }

    hipMemcpy(segmentsList_device, segmentsList.data(), sizeofSegListBytes, hipMemcpyHostToDevice);




    hipFree(segmentsList_device);
    hipFree(outputList_device);
}

void logCUDAInfo(void){
    std::cout << "=============================" << std::endl;
    std::cout << "CUDA info:" << std::endl;
    
    int driverVersion, runtimeVersion, deviceCount;

    auto e = hipRuntimeGetVersion(&runtimeVersion);
    if(e != 0){
        printf("get runtime CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    std::cout << "CUDA Runtime Version: " << runtimeVersion << std::endl;
    e = hipDriverGetVersion(&driverVersion);
    if(e != 0){
        printf("get version CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    std::cout << "CUDA Driver Version: " << driverVersion << std::endl;
    e = hipGetDeviceCount(&deviceCount);
    if(e != 0){
        printf("get device CUDA error #%d: %s\n", e, hipGetErrorString(e));
    }
    std::cout << "CUDA Device Count: " << deviceCount << std::endl;

    std::cout << "=============================" << std::endl;
}