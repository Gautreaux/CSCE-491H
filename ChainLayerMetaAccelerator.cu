#include "ChainLayerMetaAccelerator.cuh"

NVCC_G void precacheChains(char* const bitTable, const LineSegment* const segmentsList,
    const unsigned int segmentsQty, const char mode)
{
    const unsigned int mySegentIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if(mySegentIndex >= segmentsQty){
        return;
    }
    // printf("Starting index: %d\n", mySegentIndex);
    unsigned int ctr = 0;
    for(unsigned int thisRow = 0; thisRow < CEIL_DIVISION(segmentsQty, sizeof(char)*8); thisRow++){
        char c = 0;
        for(unsigned int j = 0; j < sizeof(char)*8; j++, ctr++){
            if(ctr >= segmentsQty){
                continue;
            }

            // printf("CALC: %d %d\n", mySegentIndex, ctr);

            bool b;
            switch (mode)
            {
            case 0: (b = theoretical_canMoveSegmentPair(segmentsList[mySegentIndex], segmentsList[ctr])); break;
            case 1: (b = codex_canMoveSegmentPair(segmentsList[mySegentIndex], segmentsList[ctr])); break;
            case 2: (b = current_canMoveSegmentPair(segmentsList[mySegentIndex], segmentsList[ctr])); break;
            case 3: (b = relaxed_canMoveSegmentPair(segmentsList[mySegentIndex], segmentsList[ctr])); break;
            default: b = false; break;
            }
            c |= (((b) ? 1 : 0) << j);

            // if(mySegentIndex == 0 && (ctr == 4 || ctr == 32 || ctr == 162 || ctr == 196)){
            //     printf("CUDA err: i = %d, j = %d, offset = %d, bool: %d, segmentsQty: %d, thisRow: %d,"
            //             "mySegmentIndex: %d, storeIndex: %d, ctr: %d \n",
            //             mySegentIndex, ctr, j, b, segmentsQty, thisRow, 
            //             mySegentIndex, segmentsQty*thisRow + mySegentIndex,
            //             ctr);
            // }
        }
        // int d = (int)c;
        // printf("CUDA set: %d %d\n", segmentsQty*thisRow + mySegentIndex, d);
        bitTable[segmentsQty*thisRow + mySegentIndex] = c;
    }
}

PreCache offloadPrecaching(
    const unsigned int numberPrintSegments,
    const std::vector<LineSegment>& segmentsList,
    const char mode,
    const unsigned int id, std::ostream& outStream
){
    assert(numberPrintSegments == segmentsList.size());

    const unsigned int numberRows = CEIL_DIVISION(numberPrintSegments, sizeof(char)*8);

    LineSegment* segmentsList_device;
    const unsigned int sizeofSegListBytes = sizeof(LineSegment)*numberPrintSegments;

    char* outputList_device;
    const unsigned int sizeofOutputBytes = numberRows * numberPrintSegments;

    outStream << "Attempting hipMalloc of segments/output: " << sizeofSegListBytes << " " << sizeofOutputBytes << std::endl;


    for(unsigned int i = 0; i < 2 ; i++){
        auto e = hipMalloc(
            ((i == 0) ? ((void**)&segmentsList_device) : (void**)&outputList_device),
            ((i == 0) ? sizeofSegListBytes : sizeofOutputBytes)
        );
        if(e == hipErrorOutOfMemory){
            outStream << "Device OOM?" << std::endl;
        }
        if(e != hipSuccess){
            outStream << "Error occurred in CUDA malloc: " << e << std::endl;
            outStream << "Reason: " << hipGetErrorString(e) << std::endl;
            throw e;
        };
    }

    hipMemcpy(segmentsList_device, segmentsList.data(), sizeofSegListBytes, hipMemcpyHostToDevice);
    hipMemset(outputList_device, 0, sizeofOutputBytes);

    const unsigned int threadsPerBlock = 256;
    const unsigned int numberBlocks =  CEIL_DIVISION(numberPrintSegments, threadsPerBlock);

    precacheChains<<<numberBlocks, threadsPerBlock>>>(outputList_device, segmentsList_device, numberPrintSegments, mode);

    auto e = hipDeviceSynchronize();
    if(e != hipSuccess){
        outStream << "CUDA synchronize failed with " << e << hipGetErrorString(e) << std::endl;
        throw e;
    }

    char* const outputList = (char*)malloc(sizeofOutputBytes);
    hipMemcpy(outputList, outputList_device, sizeofOutputBytes, hipMemcpyDeviceToHost);

    // for(unsigned int i = 0; i < sizeofOutputBytes; i++){
    //     int k;
    //     k = (int)(outputList[i]);
    //     printf("HOST read: %d %d\n", i, k);
    // }

    hipFree(segmentsList_device);
    hipFree(outputList_device);

    return PreCache(outputList, numberPrintSegments);
}

void logCUDAInfo(std::ostream& outStream){
    outStream << "=============================" << std::endl;
    outStream << "CUDA info:" << std::endl;
    
    int driverVersion, runtimeVersion, deviceCount;

    auto e = hipRuntimeGetVersion(&runtimeVersion);
    if(e != 0){
        outStream << "get runtime CUDA error #" << e << ": " << hipGetErrorString(e) << std::endl;
    }
    outStream << "CUDA Runtime Version: " << runtimeVersion << std::endl;
    e = hipDriverGetVersion(&driverVersion);
    if(e != 0){
        outStream << "get version CUDA error #" << e << ": " << hipGetErrorString(e) << std::endl;
    }
    outStream << "CUDA Driver Version: " << driverVersion << std::endl;
    e = hipGetDeviceCount(&deviceCount);
    if(e != 0){
        outStream << "get device CUDA error #" << e << ": " << hipGetErrorString(e) << std::endl;
    }
    outStream << "CUDA Device Count: " << deviceCount << std::endl;

    printf("!!!REMOVE ME!!!\n");
    e = hipDeviceSetLimit(hipLimitPrintfFifoSize, 1*1024*1024*1024);
    if(e != 0){
        outStream << "set limit CUDA error #" << e << ": " << hipGetErrorString(e) << std::endl;
    }
    size_t mySize;
    hipDeviceGetLimit(&mySize, hipLimitPrintfFifoSize);
    outStream << "Resolved printf size: " << mySize << std::endl;

    outStream << "=============================" << std::endl;
}

void cudaInit(void){
    
}

PreCache::PreCache(void) : c(nullptr), size(0)
{};

PreCache::PreCache(const char* const c, const unsigned int s) :
c(c), size(s)
{};

PreCache& PreCache::operator=(PreCache&& other){
    if(c != nullptr){
        free((void*)c);
        c = nullptr;
        size = 0;
    }
    size = other.size;
    c = other.c;

    other.c = nullptr;
    other.size = 0;

    return *this;
}

PreCache::~PreCache(void){
    if(c!=nullptr){
        free((void*)c);
        c = nullptr;
        size = 0;
    }
}