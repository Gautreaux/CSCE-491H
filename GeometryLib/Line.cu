#include "Line.h"

Line::Line(const Point3& point, const Slope& slope) : point(point), slope(slope){};

Line::Line(const Point3& point, const Point3& otherPoint) :
    point(point), slope(point, otherPoint)
{
    //TODO - do this one too
    // is this done though?
}

bool Line::isParallel(const Line& other) const {
    return slope == other.slope || slope == other.slope.reverse();
}

bool Line::isCollinear(const Line& other) const {
    if(!isParallel(other)){
        return false;
    }

    Line spanLine(this->point, other.point);
    return isParallel(spanLine);
}

bool Line::isOnLine(const Point3& testPoint) const {
    return isCollinear(Line(testPoint, this->slope));
}

Point3 Line::getLineIntersectPoint(const Line& other) const {
    if(isCollinear(other)){
        throw CollinearIntersectionException();
    }
    //if parallel/skew, throw no intersecton?
    //  nah, part below should catch it

    // easiest check
    if(point == other.point){
        return point;
    }
    if(other.isOnLine(this->point)){
        return point;
    }
    if(this->isOnLine(other.point)){
        return other.point;
    }

    //magic: https://math.stackexchange.com/questions/270767/find-intersection-of-two-3d-lines/271366

    Vector3 pointsVector(other.point - this->point, false);

    double cross1Mag = cross_product(other.slope, pointsVector).getMagnitude();
    double cross2Mag = cross_product(other.slope, this->slope).getMagnitude();
    if(DOUBLE_EQUAL(cross1Mag, 0) || DOUBLE_EQUAL(cross2Mag, 0)){
        throw NoIntersectionException();
    }

    Vector3 offsetVector = this->slope * (cross1Mag/cross2Mag);

    Point3 p = this->point + offsetVector;
    if(other.isOnLine(p)){
        return p;
    }else{
        p = this->point - offsetVector;
#ifdef DEBUG
        if(other.isOnLine(p) == false){
            printf("Assertion Failure inbound\n");
            std::cout << "Point p: " << p << ", Line (other): " << other << std::endl;
            std::cout << "Line (this): " << *this << ", offsetVector: " << offsetVector << std::endl;
            std::cout << "Cross1Mag: " << cross1Mag << ", cross2Mag: " << cross2Mag << std::endl;
        }
#endif
        assert(other.isOnLine(p));
        return p;
    }

}

Point3 Line::getProjectionPoint(const Point3& testPoint) const {
    Vector3 ap(testPoint - point, false);
    
    double projectionMagnitude = dot_product(ap, slope)/dot_product(slope, slope);
    return (point) + (projectionMagnitude * slope);
}

std::ostream& operator<<(std::ostream& os, const Line& line){
    os << "[" << line.getPoint() << " " << line.getSlope() << "]";
    return os;
}