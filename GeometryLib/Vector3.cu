#include "Vector3.h"

#include "math.h"

Vector3::Vector3(const double& x, const double& y, const double& z, const bool convertToUnitVector) :
    Point3(x,y,z)
{
    if(!convertToUnitVector){
        return;
    }

    double m = getMagnitude();
    if(m != 0){
        this->x /= m;
        this->y /= m;
        this->z /= m;
    }// if m == 0, everything else should also already be zero
}

Vector3::Vector3(const Point3& start, const Point3& end, const bool convertToUnitVector) :
    Point3(end - start)
{
    if(!convertToUnitVector){
        return;
    }

    double m = getMagnitude();
    if(m != 0){
        this->x /= m;
        this->y /= m;
        this->z /= m;
    }// if m == 0, everything else should also already be zero
}

Vector3::Vector3(const Point3& p, const bool convertToUnitVector) :
    Point3(p)
{
    if(!convertToUnitVector){
        return;
    }

    double m = getMagnitude();
    if(m != 0){
        this->x /= m;
        this->y /= m;
        this->z /= m;
    }// if m == 0, everything else should also already be zero
}